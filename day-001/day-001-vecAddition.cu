#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // we are still in the acceptable region of data
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float *A_h, float *B_h, float *C_h, int elements)
{
    // how many bytes do wee need to allocate?
    int size = elements * sizeof(float);

    float *A_d, *B_d, *C_d;

    // a sample of error handling for CUDA functions
    hipError_t err = hipMalloc((void **)&A_d, size);

    if (err != hipSuccess)
    {
        printf("There was an issue in memory allocation above line %d: %s", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // allocate memory on device for the arrays (for vector A, I also have error handling sample)
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    // copy from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // will do this later, the kernel...

    // copy the results to host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // free the allocated objects
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    return 0;
}