#include <hip/hip_runtime.h>

void vecAdd(float *A_h, float *B_h, float *C_h, int elements)
{
    // how many bytes do wee need to allocate?
    int size = elements * sizeof(float);

    float *A_d, *B_d, *C_d;

    // allocate memory on device for the arrays
    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    // copy from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // will do this later, the kernel...

    // copy the results to host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // free the allocated objects
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    return 0;
}