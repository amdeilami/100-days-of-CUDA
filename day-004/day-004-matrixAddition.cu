
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void matrixAdditionKernel(float *A_d, float *B_d, float *R_d, unsigned int m, unsigned int n)
{
}

__host__ void matrixAddition(float *A_h, float *B_h, float *R_h, unsigned int m, unsigned int n)
{

    unsigned int size = m * n * sizeof(float);

    float *A_d, *B_d, *R_d;
    hipError_t err = hipMalloc((void **)&A_d, size);
    if (err != hipSuccess)
    {
        printf("issue with memory allocation in device...");
        exit(1);
    }
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&R_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // calling the kernel...

    hipMemcpy(R_h, R_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(R_d);
}

__host__ int main(int argc, char *argv[])
{
    if (argc < 3)
    {
        printf("enter the dimensions of the matrices, for example: ./a.out 5 3");
    }
    unsigned int m = atoi(argv[1]);
    unsigned int n = atoi(argv[2]);

    float *A_h, *B_h, *R_h;
    unsigned int size = m * n * sizeof(float);
    A_h = (float *)malloc(size);
    B_h = (float *)malloc(size);
    R_h = (float *)malloc(size);

    srand((unsigned int)time(NULL));

    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            A_h[i * m + j] = (float)(rand() % 100000000);
            B_h[i * m + j] = (float)(rand() % 100000000);
        }
    }

    matrixAddition(A_h, B_h, R_h, m, n);
}