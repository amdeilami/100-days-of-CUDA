#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void vectorCoefficientKernel(float *A_d, float *R_d, float coefficient, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        R_d[i] = A_d[i] * coefficient;
    }
}

__host__ void vectorCoefficient(float *A_h, float *R_h, float coefficient, int n)
{
    // allocating memory on device
    int size = n * sizeof(float);

    float *A_d, *R_d;
    hipError_t err = hipMalloc((void **)&A_d, size);
    if (err != hipSuccess)
    {
        printf("There was something wrong with cuda memory allocation: %s", hipGetErrorString(err));
    }
    hipMalloc((void **)&R_d, size);

    // transfering data from host
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);

    // calling the kernel...
    vectorCoefficientKernel<<<ceil(n / 256.0), 256>>>(A_d, R_d, coefficient, n);

    // transfering result to host
    hipMemcpy(R_h, R_d, size, hipMemcpyDeviceToHost);

    // deallocating what we don't need anymore
    hipFree(A_d);
    hipFree(R_d);
}

int main(int argc, char *argv[])
{
    int n = atoi(argv[1]);
    float *A_h = (float *)malloc(n * sizeof(float));
    float *R_h = (float *)malloc(n * sizeof(float));

    srand((unsigned int)time(NULL));

    float coefficient = (float)(rand() % 10000000000);

    for (int i = 0; i < n; ++i)
    {
        A_h[i] = (float)(rand() % 1444);
        R_h[i] = 0.0;
    }

    printf("coefficient: %f\n", coefficient);

    for (int i = 0; i < n; ++i)
    {
        printf("%f ,", A_h[i]);
    }

    vectorCoefficient(A_h, R_h, coefficient, n);

    printf("\n\n");

    for (int i = 0; i < n; ++i)
    {
        printf("%f ,", R_h[i]);
    }

    printf("\n\n");

    free(A_h);
    free(R_h);

    return 0;
}